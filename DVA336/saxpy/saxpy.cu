#include "hip/hip_runtime.h"
//#include "device_launch_parameters.h"


__global__ void krnl_SAXPY(float a, float * x, int n, float * y) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
	int st = blockDim.x * gridDim.x;

    while (id < n) {
        y[id] = a * x[id] + y[id];
        id += st;
    }    
}

void computeSAXPY_CUDA(float a, float * x, int n, float * y) {
	float * dev_x;
    float * dev_y;
	
	// Allocate device memory
	hipMalloc((void **)&dev_x, n * sizeof(float));
	hipMalloc((void **)&dev_y, n * sizeof(float));
	
	// Initialize device buffers
	hipMemcpy(dev_x, x, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, n * sizeof(float), hipMemcpyHostToDevice);
		    
	// Launch kernel
	krnl_SAXPY<<<128, 256>>>(a, dev_x, n, dev_y);

	// Read back the computed result from the device
	hipMemcpy(y, dev_y, n * sizeof(float), hipMemcpyDeviceToHost);
	
	// Clean up
    hipFree(dev_x);
	hipFree(dev_y);
}
